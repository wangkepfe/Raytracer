#include "hip/hip_runtime.h"
/***********************************
 *                                 *
 *      A very n1ce raytracer      *
 *                                 *
 ***********************************/
 
// Author: Ke Wang
// Summer 2018

// C/C++
#include <iostream>
#include <time.h>

// CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// Macro and constants
#include "macro.h"
#include "constants.cuh"

// C++ header libs
#include "io_utils.h"
#include "obj_loader.h"

// CUDA header libs
#include "geometry.cuh"
#include "geometryIntersect.cuh"
#include "implicitGeometry.cuh"
#include "material.cuh"
#include "surface.cuh"
#include "meshGeometry.cuh"
#include "sceneAttributes.cuh"

__global__ void renderKernal (
    float3 *output,
    uint2 patch_offset,
    Attr attr, 
    hiprandState_t* randstates)
{
//    printf("%p %f %f %f\n", attr.meshSOA.vertices, attr.meshSOA.vertices[0].x, attr.meshSOA.vertices[0].y, attr.meshSOA.vertices[0].z);

    uint x = blockIdx.x*blockDim.x + threadIdx.x;   
    uint y = blockIdx.y*blockDim.y + threadIdx.y;
    uint idx = (PATCH_HEIGHT - y - 1) * PATCH_WIDTH + x;
    uint randStateidx = threadIdx.y * BLOCK_SIZE + threadIdx.x;

    uint realX = patch_offset.x + x;   
    uint realY = patch_offset.y + y;

    float3 deltaX = make_float3(WIDTH * camFov / HEIGHT, 0.0f, 0.0f);
    float3 deltaY = make_float3(0.0f, camFov, 0.0f);

    float3 finalColor = make_float3(0.0f, 0.0f, 0.0f);

    float3 rayDirection = normalize(camDir + deltaX * (realX * 2.0f / WIDTH - 1.0f) + deltaY * (realY * 2.0f / HEIGHT - 1.0f));

    for (uint s = 0; s < SAMPLES; s++) {//sample
        Ray currentRay {camOrig, rayDirection};

        float3 accumulativeColor = make_float3(0.0f, 0.0f, 0.0f);
        float3 colorMask = make_float3(1.0f, 1.0f, 1.0f);

        for (uint bounces = 0; hiprand_uniform(&randstates[randStateidx]) < pRussianRoulette && bounces < MAX_RAY_BOUNCE; bounces++) {//bounce
            float3 hitPoint;
            float3 normalAtHitPoint;
            bool isIntoSurface = true;

            float nearestIntersectionDistance = M_INF;
            bool hitEmptyVoidSpace = true;
            int hitObjectMaterialIdx = 0;

            for (uint objectIdx = 0; objectIdx < attr.numberOfObject; ++objectIdx) {// scene intersection
                Geometry geometry = attr.geometries[objectIdx];

                if (geometry.geometryType == SPHERE) {
                    Sphere sphere = attr.spheres[geometry.geometryIdx];
                    float distanceToObject = intersectSphereRay(sphere, currentRay);
    
                    if (distanceToObject > M_EPSILON && distanceToObject < nearestIntersectionDistance) {
                        hitEmptyVoidSpace = false;
                        nearestIntersectionDistance = distanceToObject;

                        hitPoint = currentRay.orig + currentRay.dir * distanceToObject;
                        normalAtHitPoint = getSphereNormal(hitPoint, sphere.orig, currentRay.dir, isIntoSurface);
                        hitObjectMaterialIdx = geometry.materialIdx;
                    }
                } 
                else if (geometry.geometryType == AABB) {
                    AxisAlignedBoundingBox aabb = attr.aabbs[geometry.geometryIdx];
                    float distanceToObject = intersectAABBRayBothSide(aabb, currentRay);
    
                    if (distanceToObject > M_EPSILON && distanceToObject < nearestIntersectionDistance) {
                        hitEmptyVoidSpace = false;
                        nearestIntersectionDistance = distanceToObject;

                        hitPoint = currentRay.orig + currentRay.dir * distanceToObject;
                        normalAtHitPoint = getAABBNormal(hitPoint, aabb, currentRay.dir, isIntoSurface);
                        hitObjectMaterialIdx = geometry.materialIdx;
                    }                    
                }
                else if (geometry.geometryType == MESH) {
                    Mesh mesh;
                    //printf("%p %f %f %f\n", attr.meshSOA.vertices, attr.meshSOA.vertices[1].x, attr.meshSOA.vertices[1].y, attr.meshSOA.vertices[1].z);
                    getMeshFromSOA(attr.meshSOA, geometry.geometryIdx, mesh);
                    // printf("%f %f %f\n", mesh.vertices[0].x, mesh.vertices[0].y, mesh.vertices[0].z);

                    float distanceToObject = RayMeshIntersection(normalAtHitPoint, isIntoSurface, mesh, currentRay);

                    if (distanceToObject > M_EPSILON && distanceToObject < nearestIntersectionDistance) {
                        hitEmptyVoidSpace = false;
                        nearestIntersectionDistance = distanceToObject;
                        hitPoint = currentRay.orig + currentRay.dir * distanceToObject;
                        hitObjectMaterialIdx = geometry.materialIdx;
                    }
                }
            }// end of scene intersection

            if (hitEmptyVoidSpace) {
                if (dot(currentRay.dir, sunDir) > sunSize)
                    accumulativeColor += colorMask * sunColor;
                else if (bounces == 0)
                    accumulativeColor += colorMask * skyColor;
                else
                    accumulativeColor += colorMask * mistColor;

                break;// break out of bounce
            }

            // surface
            Material material = attr.materials[hitObjectMaterialIdx];
            accumulativeColor += colorMask * material.colorEmission;

            if (material.surfaceType == DIFFUSE) {
                diffuseSurface(
                    currentRay,
                    colorMask,
                    
                    hitPoint,
                    normalAtHitPoint,
                    material.surfaceColor,

                    attr,

                    randstates,
                    randStateidx
                );
            }
            else if (material.surfaceType == SPECULAR) {
                // specularSurface(
                //     currentRay,
                //     colorMask,

                //     hitPoint,
                //     normalAtHitPoint,
                //     material.surfaceColor,
                //     randstates,
                //     randStateidx
                // );
            }
            else if (material.surfaceType == MIRROR) {
                mirrorSurface(
                    currentRay,
                    colorMask,

                    hitPoint,
                    normalAtHitPoint,
                    material.surfaceColor
                );
            }
            else if (material.surfaceType == TRANSPARENT) {
                transparentSurface(
                    currentRay,
                    colorMask,

                    isIntoSurface,
                    hitPoint,
                    normalAtHitPoint,
                    randstates,
                    randStateidx
                );
            }
            
        }//end of bounce
        finalColor += accumulativeColor / SAMPLES;

    }//end of sample

    output[idx] = make_float3(clamp(finalColor.x, 0.0f, 1.0f), clamp(finalColor.y, 0.0f, 1.0f), clamp(finalColor.z, 0.0f, 1.0f));
}

__global__ void initRandStates(uint seed, hiprandState_t* randstates) {
    uint idx = threadIdx.y * BLOCK_SIZE + threadIdx.x;
    hiprand_init(seed, idx, 0, &randstates[idx]);
}

int main(){
    // define dim
    dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);   
    dim3 grid(PATCH_WIDTH / block.x, PATCH_HEIGHT / block.y, 1);

    // rand states
    hiprandState_t* randstates_d;
    hipMalloc((void**) &randstates_d, BLOCK_SIZE * BLOCK_SIZE * sizeof(hiprandState_t));
    initRandStates<<<1, block>>>(time(NULL), randstates_d);

    // build the scene
    AxisAlignedBoundingBox myHouseAABB = AxisAlignedBoundingBox {float3{-80.0f, -40.0f, -10.0f},float3{80.0f, 80.0f, 250.0f}};
    Sphere sphereOnTheCeiling = Sphere {float3{0.0f, 90.0f, 170.0f} ,20.0f};
    Sphere sphereOnTheCeiling2 = Sphere {float3{0.0f, 90.0f, 100.0f} ,20.0f};
    Sphere sphereOnTheGround = Sphere {float3{50.0f, -30.0f, 120.0f} ,10.0f};

    Mesh testMesh = loadObj("cone.obj");

    scaleMesh(testMesh, float3{50.0f, 50.0f, 50.0f});
    rotateMesh(testMesh, float3{0.0f, M_PI, 0.0f});
    translateMesh(testMesh, float3{0.0f, 10.0f, 190.0f});
    
    Material whiteDiffuse {DIFFUSE, float3{0.0f, 0.0f, 0.0f}, float3{0.75f, 0.75f, 0.75f}};
    Material redDiffuse {DIFFUSE, float3{0.0f, 0.0f, 0.0f}, float3{1.0f, 0.1f, 0.1f}};
    Material whiteLight {DIFFUSE, float3{2.0f, 2.0f, 2.0f}, float3{0.75f, 0.75f, 0.75f}};

    Sphere spheres[] {
        sphereOnTheCeiling,
        sphereOnTheCeiling2,
        sphereOnTheGround
    };

    AxisAlignedBoundingBox aabbs[] {
        myHouseAABB
    };

    //Mesh *meshes;
    uint meshNum = 1;
    Mesh meshes[] {
        testMesh
    };

    Material materials[] {
        whiteDiffuse,
        whiteLight,
        redDiffuse
    };

    Geometry myHouse {AABB, 0, 0};
    Geometry myCeilingLight {SPHERE, 0, 1};
    Geometry myCeilingLight2 {SPHERE, 1, 1};
    Geometry myFLoorLight {SPHERE, 2, 1};
    Geometry myNiceMesh {MESH, 0, 2};

    Geometry geometries[] {
        myHouse,
        myCeilingLight,
        myCeilingLight2,
        myFLoorLight,
        myNiceMesh
    };
    uint geometryNum = sizeof(geometries) / sizeof(Geometry);

    uint lightIndices[] = {1, 2, 3};
    uint lightNum = sizeof(lightIndices) / sizeof(uint);

    // copy data to cuda
    CUDA_MALLOC_MEMCPY_HOST_TO_DEVICE(Sphere, spheres_d, spheres, sizeof(spheres))
    CUDA_MALLOC_MEMCPY_HOST_TO_DEVICE(AxisAlignedBoundingBox, aabbs_d, aabbs, sizeof(aabbs))

    Meshes_SOA meshSOA = convertMeshAOSToSOA(meshes, meshNum);

    CUDA_MALLOC_MEMCPY_HOST_TO_DEVICE(Vertex, vertices_d, meshSOA.vertices, meshSOA.vertexNum * sizeof(Vertex))
    CUDA_MALLOC_MEMCPY_HOST_TO_DEVICE(Face, faces_d, meshSOA.faces, meshSOA.faceNum * sizeof(Face))
    CUDA_MALLOC_MEMCPY_HOST_TO_DEVICE(Mesh_IndexOnly, meshes_d, meshSOA.meshes, meshSOA.meshNum * sizeof(Mesh_IndexOnly))

    Meshes_SOA meshSOA_d;
    meshSOA_d.vertices = vertices_d;
    meshSOA_d.faces = faces_d;
    meshSOA_d.meshes = meshes_d;

    CUDA_MALLOC_MEMCPY_HOST_TO_DEVICE(Material, materials_d, materials, sizeof(materials))
    CUDA_MALLOC_MEMCPY_HOST_TO_DEVICE(Geometry, geometries_d, geometries, sizeof(geometries))

    CUDA_MALLOC_MEMCPY_HOST_TO_DEVICE(uint, lightIndices_d, lightIndices, sizeof(lightIndices))

    Attr attr {
        geometryNum, 
        lightNum,

        spheres_d, 
        aabbs_d,

        meshSOA_d,

        materials_d, 
        geometries_d,

        lightIndices_d
    };

    // start rendering
    float3* output = new float3[WIDTH * HEIGHT];
    for (uint i = 0; i < WIDTH * HEIGHT; ++i) output[i] = make_float3(0.0f, 0.0f, 0.0f);
    float3* output_patch = new float3[PATCH_WIDTH * PATCH_HEIGHT];
    float3* output_d;
    hipMalloc(&output_d, PATCH_WIDTH * PATCH_HEIGHT * sizeof(float3));

    uint progressRecord = 0;
    printf("Rendering...0%%\n");

    for (uint patch_i = 0; patch_i < PATCH_NUM_X; ++patch_i) {
        for (uint patch_j = 0; patch_j < PATCH_NUM_Y; ++patch_j) {
            for (uint kernalLoop_i = 0; kernalLoop_i < KERNAL_LOOP; ++kernalLoop_i) {   
                renderKernal <<< grid, block >>> (
                    output_d,
                    uint2{patch_i*PATCH_WIDTH, (PATCH_NUM_Y - patch_j - 1)*PATCH_HEIGHT},
                    attr,
                    randstates_d);
                hipMemcpy(output_patch, output_d, PATCH_WIDTH * PATCH_HEIGHT * sizeof(float3), hipMemcpyDeviceToHost);
                
                for (uint i = 0; i < PATCH_WIDTH; ++i) {
                    for (uint j = 0; j < PATCH_HEIGHT; ++j) {
                        output[(patch_j*PATCH_HEIGHT + j) * WIDTH + patch_i*PATCH_WIDTH + i] += output_patch[j * PATCH_WIDTH + i];
                    }
                }  

                uint progressPercent = ((patch_i * PATCH_NUM_Y + patch_j) * KERNAL_LOOP + kernalLoop_i) * 10 / PATCH_NUM_X / PATCH_NUM_Y / KERNAL_LOOP;
                if (progressRecord != progressPercent) {
                    progressRecord = progressPercent;
                    printf("Rendering...%d0%%\n", progressRecord);
                }
            }
        }
    }

    for (uint i = 0; i < WIDTH * HEIGHT; ++i) output[i] /= KERNAL_LOOP;

    printf("Rendering...100%%\n");
    printf("Done!\n");

    // output
    writeToPPM("result.ppm", WIDTH, HEIGHT, output);

    // clean
    hipFree(spheres_d); 
    hipFree(aabbs_d);
    
    hipFree(vertices_d);
    hipFree(faces_d);
    hipFree(meshes_d);

    hipFree(materials_d);
    hipFree(geometries_d);

    hipFree(output_d);  

    hipFree(randstates_d);

    deleteMeshSOA(meshSOA);
    delete[] output;
    delete[] output_patch;
}