#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <>
#include <time.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "io_utils.h"
#include "macro.h"

#include "geometryIntersect.cuh"
#include "surface.cuh"
#include "implicitGeometry.cuh"

enum{
    EMISSION_ONLY,
    DIFFUSE,
    SPECULAR,
    REFRACTION,
};

enum{
    IMPLICIT_SPHERE,
    IMPLICIT_AABB,
};

struct Material{
    int surfaceType;
    float3 colorEmission;
    float3 surfaceColor;
};

struct Geometry{
    int geometryType;
    int geometryIdx;
    int materialIdx;
};

struct Attr{
    int numberOfObject;
    Sphere* spheres;
    AABB* aabbs;
    Material* materials;
    Geometry* geometries;
};

__constant__ float sunSize = 0.97f;
__constant__ float3 sunDir{0.0f, 1.0f, 0.0f};
__constant__ float3 sunColor{1.0f, 0.875f, 0.75f};
__constant__ float3 skyColor{0.5f, 0.8f, 0.9f};
__constant__ float3 mistColor{0.02f, 0.02f, 0.02f};

__constant__ float3 camOrig{0.0f, 0.0f, 0.0f};
__constant__ float3 camDir{0.0f, 0.0f, 1.0f};
__constant__ float camFov = 0.5135f;

__global__ void renderKernal (float3 *output, Attr attr, hiprandState_t* randstates) {
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;   
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int idx = (HEIGHT - y - 1) * WIDTH + x; 

    float3 deltaX = make_float3(WIDTH * camFov / HEIGHT, 0.0f, 0.0f);
    float3 deltaY = make_float3(0.0f, camFov, 0.0f);

    float3 finalColor = make_float3(0.0f, 0.0f, 0.0f);

    float3 rayDirection = normalize(camDir + deltaX * (x * 2.0f / WIDTH - 1.0f) + deltaY * (y * 2.0f / HEIGHT - 1.0f));

    for (int s = 0; s < SAMPLES; s++) {//sample
        Ray currentRay {camOrig, rayDirection};

        float3 accumulativeColor = make_float3(0.0f, 0.0f, 0.0f);
        float3 colorMask = make_float3(1.0f, 1.0f, 1.0f);

        for (int bounces = 0; bounces < RAY_BOUNCE; bounces++) {//bounce
            float3 hitPoint;
            float3 normalAtHitPoint;
            bool isIntoSurface = true;

            float nearestIntersectionDistance = M_INF;
            bool hitEmptyVoidSpace = true;
            int hitObjectMaterialIdx = 0;

            for (int objectIdx = 0; objectIdx < attr.numberOfObject; ++objectIdx) {// scene intersection
                Geometry geometry = attr.geometries[objectIdx];

                if (geometry.geometryType == IMPLICIT_SPHERE) {
                    Sphere sphere = attr.spheres[geometry.geometryIdx];
                    float distanceCameraToObject = intersectSphereRay(sphere, currentRay);
    
                    if (distanceCameraToObject > 0.001f && distanceCameraToObject < nearestIntersectionDistance) {
                        hitEmptyVoidSpace = false;
                        nearestIntersectionDistance = distanceCameraToObject;

                        hitPoint = currentRay.orig + currentRay.dir * distanceCameraToObject;
                        normalAtHitPoint = getSphereNormal(hitPoint, sphere.orig, currentRay.dir, isIntoSurface);
                        hitObjectMaterialIdx = geometry.materialIdx;
                    }
                } 
                else if (geometry.geometryType == IMPLICIT_AABB) {
                    AABB aabb = attr.aabbs[geometry.geometryIdx];
                    float distanceCameraToObject = intersectAABBRay(aabb, currentRay);
    
                    if (distanceCameraToObject > 0.001f && distanceCameraToObject < nearestIntersectionDistance) {
                        hitEmptyVoidSpace = false;
                        nearestIntersectionDistance = distanceCameraToObject;

                        hitPoint = currentRay.orig + currentRay.dir * distanceCameraToObject;
                        normalAtHitPoint = getAABBNormal(hitPoint, aabb, currentRay.dir);
                        hitObjectMaterialIdx = geometry.materialIdx;
                    }                    
                }
            }// end of scene intersection

            if (hitEmptyVoidSpace) {
                if (dot(currentRay.dir, sunDir) > sunSize) // sun
                    accumulativeColor += colorMask * sunColor;
                else if (bounces == 0) // sky
                    accumulativeColor += colorMask * skyColor;
                else // mist
                    accumulativeColor += colorMask * mistColor;

                break;// break out of bounce
            }

            // surface
            Material material = attr.materials[hitObjectMaterialIdx];
            accumulativeColor += colorMask * material.colorEmission;

            if (material.surfaceType == DIFFUSE) {
                diffuseSurface(
                    currentRay,
                    colorMask,
                    hitPoint,
                    normalAtHitPoint,
                    material.surfaceColor,
                    randstates,
                    idx
                );
            }
            else if (material.surfaceType == SPECULAR) {
                specularSurface(
                    currentRay,
                    colorMask,
                    hitPoint,
                    normalAtHitPoint,
                    material.surfaceColor
                );
            }
            else if (material.surfaceType == REFRACTION) {
                refractionSurface(
                    currentRay,
                    colorMask,
                    isIntoSurface,
                    hitPoint,
                    normalAtHitPoint,
                    material.surfaceColor,
                    randstates,
                    idx
                );
            }
        }//end of bounce
        finalColor += accumulativeColor / SAMPLES;

    }//end of sample

    output[idx] = make_float3(clamp(finalColor.x, 0.0f, 1.0f), clamp(finalColor.y, 0.0f, 1.0f), clamp(finalColor.z, 0.0f, 1.0f));
}

__global__ void initRandStates(unsigned int seed, hiprandState_t* randstates) {
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;   
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int idx = (HEIGHT - y - 1) * WIDTH + x;

    hiprand_init(seed, idx, 0, &randstates[idx]);
}

int main(){
    // define dim
    dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);   
    dim3 grid(WIDTH / block.x, HEIGHT / block.y, 1);

    // rand states
    hiprandState_t* randstates;
    hipMalloc((void**) &randstates, NUM_BLOCKS * sizeof(hiprandState_t));

    // run
    initRandStates<<<grid, block>>>(time(NULL), randstates);

    // scene
    Sphere spheres[] {
        {float3{0.0f, -25.0f, 100.0f} ,15.0f},
        {float3{40.0f, -25.0f, 90.0f}, 15.0f},
        {float3{30.0f, -20.0f, 160.0f}, 20.0f},
        {float3{-50.0f, -25.0f, 90.0f}, 15.0f}
    };

    AABB aabbs[] {
        {float3{-M_INF, -50.0f, -M_INF},float3{M_INF, -40.0f, M_INF}},
        {float3{-50.0f, -40.0f, 150.0f},float3{-30.0f, -20.0f, 130.0f}}
    };

    Material materials[] {
        {DIFFUSE, float3{1.0f, 1.0f, 1.0f}, float3{0.75f, 0.75f, 0.75f}},
        {DIFFUSE, float3{0.0f, 0.0f, 0.0f}, float3{0.75f, 0.75f, 0.75f}},
        {DIFFUSE, float3{0.0f, 0.0f, 0.0f}, float3{0.9f, 0.2f, 0.1f}},
        {DIFFUSE, float3{0.0f, 0.0f, 0.0f}, float3{0.1f, 0.2f, 0.9f}},
        {SPECULAR, float3{0.0f, 0.0f, 0.0f}, float3{0.1f, 0.9f, 0.1f}},
        {REFRACTION, float3{0.0f, 0.0f, 0.0f}, float3{1.0f, 1.0f, 1.0f}}
    };

    Geometry geometries[] {
        {IMPLICIT_AABB, 0, 1},
        {IMPLICIT_AABB, 1, 2},
        {IMPLICIT_SPHERE, 0, 0},
        {IMPLICIT_SPHERE, 1, 3},
        {IMPLICIT_SPHERE, 2, 4},
        {IMPLICIT_SPHERE, 3, 5}
    };

    Sphere* spheres_d;
    AABB* aabbs_d;
    Material* materials_d;
    Geometry* geometries_d;

    hipMalloc(&spheres_d, sizeof(spheres));
    hipMalloc(&aabbs_d, sizeof(aabbs));
    hipMalloc(&materials_d, sizeof(materials));
    hipMalloc(&geometries_d, sizeof(geometries));

    hipMemcpy(spheres_d, spheres, sizeof(spheres), hipMemcpyHostToDevice);
    hipMemcpy(aabbs_d, aabbs, sizeof(aabbs), hipMemcpyHostToDevice);
    hipMemcpy(materials_d, materials, sizeof(materials), hipMemcpyHostToDevice);
    hipMemcpy(geometries_d, geometries, sizeof(geometries), hipMemcpyHostToDevice);

    Attr attr {
        sizeof(geometries) / sizeof(Geometry), 
        spheres_d, 
        aabbs_d,
        materials_d, 
        geometries_d
    };
    
    // run
    float3* output = new float3[WIDTH * HEIGHT];
    float3* output_h = new float3[WIDTH * HEIGHT];
    float3* output_d;
    hipMalloc(&output_d, WIDTH * HEIGHT * sizeof(float3));

    for (int i = 0; i < KERNAL_LOOP; ++i) {   
        renderKernal <<< grid, block >>> (output_d, attr, randstates);
        hipMemcpy(output_h, output_d, WIDTH * HEIGHT * sizeof(float3), hipMemcpyDeviceToHost);
        if (i == 0) {
            for (int j = 0; j < WIDTH * HEIGHT; ++j) {
                output[j] = output_h[j];
            }
        } else {
            for (int j = 0; j < WIDTH * HEIGHT; ++j) {
                output[j] += output_h[j];
                output[j] /= 2;
            }
        }
        int progressPercent = i * 100 / KERNAL_LOOP;
        if (progressPercent % 10 == 0)
            printf("Rendering...%d%%\n", progressPercent);
    }
    printf("Done!\n");
    
    // output
    writeToPPM("result.ppm", WIDTH, HEIGHT, output);

    // clean
    hipFree(spheres_d); 
    hipFree(aabbs_d);  
    hipFree(materials_d);
    hipFree(geometries_d);

    hipFree(output_d);  
    hipFree(randstates);

    delete[] output;
    delete[] output_h;
}